#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "kernel.h"

__global__ void _gpu_pairwise_mul(long *a, long *b, long *res) {
    int i = threadIdx.x;
    res[i] = a[i] * b[i];
}

extern "C" void gpu_pairwise_mul(long *a, long *b, long *res, int n_elems) {
    size_t size = n_elems * sizeof(long);

    long *cuda_a = 0;
    long *cuda_b = 0;
    long *cuda_res = 0;

    hipMalloc(&cuda_a, size);
    hipMalloc(&cuda_b, size);
    hipMalloc(&cuda_res, size);

    hipMemcpy(cuda_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, size, hipMemcpyHostToDevice);

    _gpu_pairwise_mul<<<1, n_elems>>>(cuda_a, cuda_b, cuda_res);
    hipMemcpy(res, cuda_res, size, hipMemcpyDeviceToHost);

    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_res);
}
